#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

#define BASE_TYPE double
#define M_PI  3.141592653
#define n 1e8

__global__ void sinMass(BASE_TYPE* A, int arraySize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < arraySize)
        A[index] = sin((BASE_TYPE)((index % 360) * M_PI / 180));
}

__global__ void sinfMass(BASE_TYPE* A, int arraySize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < arraySize)
        A[index] = sinf((BASE_TYPE)((index % 360) * M_PI / 180));
}

__global__ void sinCudaMass(BASE_TYPE* A, int arraySize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < arraySize)
        A[index] = __sinf((BASE_TYPE)((index % 360) * M_PI / 180));
}

double calcSinError(void(*fn)(BASE_TYPE*, int), BASE_TYPE* arr_cpu, BASE_TYPE* arr_gpu,
    unsigned gridSize, unsigned blockSize);

int main()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    if (devCount == 0)
    {
        std::cout << "No CUDA devices detected" << std::endl;
        return 0;
    }
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int blockSize = devProp.maxThreadsPerBlock;
    
    unsigned gridSize = (n + blockSize - 1) / blockSize;

    BASE_TYPE* arr_gpu;
    BASE_TYPE* arr_cpu = new BASE_TYPE[n];

    hipMalloc(&arr_gpu, n * sizeof(BASE_TYPE));

    
    double sinErr = calcSinError(sinMass, arr_cpu, arr_gpu, gridSize, blockSize);
    std::cout << "Sin error: " << sinErr << std::endl;

    double sinfErr = calcSinError(sinfMass, arr_cpu, arr_gpu, gridSize, blockSize);
    std::cout << "Sinf error: " << sinfErr << std::endl;

    double sinfCudaErr = calcSinError(sinCudaMass, arr_cpu, arr_gpu, gridSize, blockSize);
    std::cout << "Sinf CUDA error: " << sinfCudaErr << std::endl;

    delete[] arr_cpu;
    hipFree(arr_gpu);

    return 0;
}

double calcSinError(void(*fn)(BASE_TYPE*, int), BASE_TYPE* arr_cpu, BASE_TYPE* arr_gpu,
    unsigned gridSize, unsigned blockSize)
{
    (*fn) << < gridSize, blockSize >> > (arr_gpu, n);

    hipMemcpy(arr_cpu, arr_gpu, n * sizeof(BASE_TYPE), hipMemcpyDeviceToHost);
    double err = 0;
    for (int i = 0; i < n; i++) {
        err += fabs(sin((i % 360) * M_PI / 180.0) - arr_cpu[i]);
    }
    return err /= n;

}